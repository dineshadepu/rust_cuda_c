#include "hip/hip_runtime.h"
/* File:     vec_add.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 * Compile:  nvcc [-g] [-G] -o vec_add vec_add.cu
 * Run:      ./vec_add
 */

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void Vec_add(double x[], double y[], double z[], int n) {
  int thread_id = threadIdx.x;
  if (thread_id < n){
    z[thread_id] = x[thread_id] + y[thread_id];
  }
}


extern "C" void axpb_cpp_cuda(double h_x[], double h_y[], double h_z[], int n) {
  double *d_x, *d_y, *d_z;
  size_t size;

  /* Define vector length */
  size = n*sizeof(double);

  // Print original vectors.
  printf("h_x = ");
  for (int i = 0; i < n; i++){
    printf("%f ", h_x[i]);
  }
  printf("\n\n");
  printf("h_y = ");
  for (int i = 0; i < n; i++){
    printf("%.1f ", h_y[i]);
  }
  printf("\n\n");


  /* Allocate vectors in device memory */
  hipMalloc(&d_x, size);
  hipMalloc(&d_y, size);
  hipMalloc(&d_z, size);

  /* Copy vectors from host memory to device memory */
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

  /* Kernel Call */
  Vec_add<<<1,1000>>>(d_x, d_y, d_z, n);

  hipDeviceSynchronize();
  hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);
  printf("The sum is: \n");
  for (int i = 0; i < n; i++){
    printf("%.1f ", h_z[i]);
  }
  printf("\n");


  /* Free device memory */
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  /* Free host memory */
  /* free(h_x); */
  /* free(h_y); */
  /* free(h_z); */


}  /* main */
